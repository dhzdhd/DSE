#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdlib.h>
#include <stdio.h>

#define SIZE 5

__global__ void rowKernel(int* c, const int* a, const int* b)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < SIZE) {
        for (int j = 0; j < SIZE; j++) {
            int buf = 0;
            for (int k = 0; k < SIZE; k++) {
                buf += a[i * SIZE + k] * b[k * SIZE + j];
            }
            c[i * SIZE + j] = buf;
        }
    }
}

__global__ void colKernel(int* c, const int* a, const int* b)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (j < SIZE) {
        for (int i = 0; i < SIZE; i++) {
            int buf = 0;
            for (int k = 0; k < SIZE; k++) {
                buf += a[i * SIZE + k] * b[k * SIZE + j];
            }
            c[i * SIZE + j] = buf;
        }
    }
}

__global__ void allKernel(int* c, const int* a, const int* b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int buf = 0;

    if (i < SIZE && j < SIZE) {
        for (int k = 0; k < SIZE; k++) {
             buf += a[i * SIZE + k] * b[k * SIZE + j];
        }
        c[i * SIZE + j] = buf;
    }
}

int main()
{

    int* a = (int*)calloc(SIZE * SIZE, sizeof(int));
    int* b = (int*)calloc(SIZE * SIZE, sizeof(int));
    int* c = (int*)calloc(SIZE * SIZE, sizeof(int));

    for (int i = 0; i < SIZE * SIZE; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;

        printf("%d,%d  ", a[i], b[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }
    printf("\n");

    int* dev_a, * dev_b, * dev_c;
    hipMalloc((int**)&dev_a, SIZE * SIZE * sizeof(int));
    hipMalloc((int**)&dev_b, SIZE * SIZE * sizeof(int));
    hipMalloc((int**)&dev_c, SIZE * SIZE * sizeof(int));

    hipMemcpy(dev_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    dim3 row_threads(SIZE, SIZE, 1);
    dim3 row_blocks(1, 1, 1);

    rowKernel << <row_blocks, row_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    printf("\n");

    dim3 col_threads(SIZE, SIZE, 1);
    dim3 col_blocks(1, 1, 1);

    colKernel << <col_blocks, col_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    printf("\n");

    dim3 all_threads(SIZE, SIZE, 1);
    dim3 all_blocks(1, 1, 1);

    allKernel << <all_blocks, all_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    return 0;
}

