#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdlib.h>
#include <stdio.h>

#define SIZE 5

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
        
    c[i * SIZE + j] = a[i * SIZE + j] + b[i * SIZE + j];
}

int main()
{
    
    int* a = (int*)calloc(SIZE * SIZE, sizeof(int));
    int* b = (int*)calloc(SIZE * SIZE, sizeof(int));
    int* c = (int*)calloc(SIZE * SIZE, sizeof(int));
    
    for (int i = 0; i < SIZE * SIZE; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;

        printf("%d,%d  ", a[i], b[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }
    printf("\n");
    
    int* dev_a, * dev_b, * dev_c;
    hipMalloc((int**)&dev_a, SIZE * SIZE * sizeof(int));
    hipMalloc((int**)&dev_b, SIZE * SIZE * sizeof(int));
    hipMalloc((int**)&dev_c, SIZE * SIZE * sizeof(int));

    hipMemcpy(dev_a, a, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    dim3 row_threads(1, SIZE, 1);
    dim3 row_blocks(1, SIZE, 1);

    addKernel << <row_blocks, row_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    printf("\n");

    dim3 col_threads(SIZE, 1, 1);
    dim3 col_blocks(SIZE, 1, 1);

    addKernel << <col_blocks, col_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    printf("\n");

    dim3 all_threads(SIZE, SIZE, 1);
    dim3 all_blocks(1, 1, 1);

    addKernel << <all_blocks, all_threads >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE * SIZE; i++) {
        printf("%d  ", c[i]);
        if ((i + 1) % SIZE == 0 && i != 0) {
            printf("\n");
        }
    }

    return 0;
}

