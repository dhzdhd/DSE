#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <math.h>

__global__ void sine_gpu(double* a, double* b)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	b[index] = __sinf(a[index]);
}

int main()
{
	int count;
	std::cout << "Enter length of array\n";
	std::cin >> count;

	double* host_a = new double[count];
	double* host_b = new double[count];

	for (int i = 0; i < count; i++) {
		*(host_b + i) = 0.0;

		int r = (double)rand() / 100;
		*(host_a + i) = r;
	}

	double * device_a, * device_b;

	const int size = count * sizeof(int);

	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMemcpy(
		device_a, host_a,
		size,
		hipMemcpyHostToDevice
	);

	int blocks = count / 100 + 1;
	int threads = 100;
	sine_gpu << < blocks, threads >> > (device_a, device_b);

	hipMemcpy(
		host_b, device_b,
		size,
		hipMemcpyDeviceToHost
	);

	for (auto i = 0; i < count; i++)
	{
		std::cout << "sine " << host_a[i] << " = " << host_b[i] << "\n";
	}
	getchar();
	return 0;
}
