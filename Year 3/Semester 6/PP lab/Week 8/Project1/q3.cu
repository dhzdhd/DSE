#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>

__global__ void conv_gpu(int* a, int* b, int* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	int count, mask;
	std::cout << "Enter length of array and mask\n";
	std::cin >> count >> mask;

	int* host_a = new int[count];
	int* host_b = new int[mask];
	int* host_c = new int[count];

	for (int i = 0; i < count; i++) {
		*(host_c + i) = 0;

		int r = rand() / 100;
		*(host_a + i) = r;
	}

	for (int i = 0; i < mask; i++) {
		int r = rand() / 100;
		*(host_b + i) = r;
	}

	int* device_a, * device_b, * device_c;

	const int size = count * sizeof(int);

	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMalloc(&device_c, size);
	hipMemcpy(
		device_a, host_a,
		size,
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		device_b, host_b,
		size,
		hipMemcpyHostToDevice
	);

	int blocks = count / 100 + 1;
	int threads = 100;
	conv_gpu << < blocks, threads >> > (device_a, device_b, device_c);

	hipMemcpy(
		host_c, device_c,
		size,
		hipMemcpyDeviceToHost
	);

	for (auto i = 0; i < count; i++)
	{
		std::cout << host_a[i] << " conv " << host_b[i] << " = " << host_c[i] << "\n";
	}
	getchar();
	return 0;
}
