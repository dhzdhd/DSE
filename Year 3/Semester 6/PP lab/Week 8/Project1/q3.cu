#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>

__global__ void conv_gpu(int* a, int* b, int* c, int count, int mask)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int k;

	if (index < count) {
		int pValue = 0;
		// Convolution operation
		for (k = 0; k < mask; ++k) {
			int maskIndex = mask - 1 - k;
			int nIndex = index - (mask / 2) + k;
			if (nIndex >= 0 && nIndex < count) {
				pValue += a[nIndex] * b[maskIndex];
			}
		}
		c[index] = pValue;
	}
}

int main()
{
	int count, mask;
	std::cout << "Enter length of array and mask\n";
	std::cin >> count >> mask;

	int* host_a = new int[count];
	int* host_b = new int[mask];
	int* host_c = new int[count];

	for (int i = 0; i < count; i++) {
		*(host_c + i) = 0;

		int r = rand() / 100;
		*(host_a + i) = r;
	}

	for (int i = 0; i < mask; i++) {
		int r = rand() / 100;
		*(host_b + i) = r;
	}

	int* device_a, * device_b, * device_c;

	const int size = count * sizeof(int);

	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMalloc(&device_c, size);
	hipMemcpy(
		device_a, host_a,
		size,
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		device_b, host_b,
		size,
		hipMemcpyHostToDevice
	);

	int blocks = count / 100 + 1;
	int threads = 100;
	conv_gpu << < blocks, threads >> > (device_a, device_b, device_c, count, mask);

	hipMemcpy(
		host_c, device_c,
		size,
		hipMemcpyDeviceToHost
	);

	for (auto i = 0; i < count; i++)
	{
		std::cout << host_c[i] << "\n";
	}
	getchar();
	return 0;
}
