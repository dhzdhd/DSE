#include "hip/hip_runtime.h"

#include <iostream>
#include <stdlib.h>

__global__ void add_arrays_gpu(int* a, int* b, int* c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

int main()
{
	int count;
	std::cout << "Enter length\n";
	std::cin >> count;

	int* host_a = new int[count];
	int* host_b = new int[count];
	int* host_c = new int[count];

	for (int i = 0; i < count; i++) {
		*(host_c + i) = 0;

		int r = rand() / 100;
		*(host_a + i) = r;
		r = rand() / 100;
		*(host_b + i) = r;
	}

	int* device_a, * device_b, * device_c;

	const int size = count * sizeof(int);

	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMalloc(&device_c, size);
	hipMemcpy(
		device_a, host_a,
		size,
		hipMemcpyHostToDevice
	);
	hipMemcpy(
		device_b, host_b,
		size,
		hipMemcpyHostToDevice
	);

	add_arrays_gpu <<< count / 256 + 1, 256 >>> (device_a, device_b, device_c);

	hipMemcpy(
		host_c, device_c,
		size,
		hipMemcpyDeviceToHost
	);

	for (auto i = 0; i < count; i++)
	{
		std::cout << host_a[i] << "+" << host_b[i] << "=" <<host_c[i] << "\n";
	}
	getchar();
	return 0;
}