#include "hip/hip_runtime.h"

#include <iostream>

__global__ void add_arrays_gpu(int *a, int *b, int *c)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main()
{
	int count;
	std::cout << "Enter length\n";
	std::cin >> count;

	std::cout << "Enter elements\n";
	int *host_a = new int[count];
	int *host_b = new int[count];
	int *host_c = new int[count];

	for (int i = 0; i < count; i++)
	{
		*(host_c + i) = 0;
		std::cin >> *(host_a + i);
		std::cin >> *(host_b + i);
	}

	int *device_a, *device_b, *device_c;

	const int size = count * sizeof(int);

	hipMalloc(&device_a, size);
	hipMalloc(&device_b, size);
	hipMalloc(&device_c, size);
	hipMemcpy(
		device_a, host_a,
		size,
		hipMemcpyHostToDevice);
	hipMemcpy(
		device_b, host_b,
		size,
		hipMemcpyHostToDevice);

	add_arrays_gpu<<<1, count>>>(device_a, device_b, device_c);

	hipMemcpy(
		host_c, device_c,
		size,
		hipMemcpyDeviceToHost);

	for (auto i = 0; i < count; i++)
	{
		std::cout << host_c[i] << " ";
	}
	getchar();
	return 0;
}
