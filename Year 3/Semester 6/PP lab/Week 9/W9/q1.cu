#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>

#define MAX_SENTENCE_LENGTH 100
#define WORD_LENGTH 20

__global__ void wordCount(char* sentence, char* word, int* result, int SentenceLength, int wordLength) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int localCount = 0;

    for (int i = index; i <= SentenceLength - wordLength; i += stride) {
        bool wordFound = true;
        for (int j = 0; j < wordLength; j++) {
            if (sentence[i + j] != word[j]) {
                wordFound = false;
                break;
            }
        }
        if (wordFound) {
            localCount++;
        }
    }

    atomicAdd(result, localCount);
}

int main() {
    char sentence[MAX_SENTENCE_LENGTH] = "hello world hello hi hi hi hello";
    char word[WORD_LENGTH] = "hello";
    
    int result = 0;
    int senLen = strlen(sentence);
    int worLen = strlen(word);

    char* d_sentence, * d_word;
    int* d_result;

    hipMalloc((void**)&d_sentence, MAX_SENTENCE_LENGTH * sizeof(char));
    hipMalloc((void**)&d_word, WORD_LENGTH * sizeof(char));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_sentence, sentence, MAX_SENTENCE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);

    wordCount << <(MAX_SENTENCE_LENGTH + 255) / 256, 256 >> > (d_sentence, d_word, d_result, senLen, worLen);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word, result);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_result);

    return 0;
}