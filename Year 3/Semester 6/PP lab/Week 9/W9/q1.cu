#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudart_platform.h"
#include ""
#include "device_atomic_functions.h"
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <malloc.h>

__global__ void count(char* str, char* pattern, int* res, int len_s, int len_p) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int extent = len_s - len_p;

	bool isPresent = true;
	//if (x <= extent) {
		for (int i = 0; i < len_p; i++) {
			if (str[x + i] != pattern[i]) {
				isPresent = false;
				break;
			}
		}

		if (isPresent) {
			atomicAdd(res, 1);
		}
	//}
}

int main() {
	int len_s, len_p;

	printf("Enter size of string and pattern\n");
	scanf("%d%d", &len_s, &len_p);
	getchar();

	char* str = (char*)malloc(len_s * sizeof(char));
	char* pattern = (char*)malloc(len_p * sizeof(char));
	int res = 0;

	printf("Enter string\n");
	gets_s(str, len_s);

	printf("Enter pattern\n");
	gets_s(str, len_p);

	char* device_str, * device_pattern;
	int* device_res;

	hipMalloc((void**)&device_str, len_s * sizeof(char));
	hipMalloc((void**)&device_pattern, len_p * sizeof(char));
	hipMalloc((void**)&device_res, sizeof(int));

	hipMemcpy(device_str, str, len_s * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(device_pattern, pattern, len_p * sizeof(char), hipMemcpyHostToDevice);
	//hipMemcpy(device_res, &res, sizeof(int), hipMemcpyHostToDevice);

	count << <1, len_s - len_p >> > (device_str, device_pattern, device_res, len_s, len_p);

	hipMemcpy(&res, device_res, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d", res);

	return 0;
}
