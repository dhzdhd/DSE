#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudart_platform.h"
#include ""
#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <malloc.h>

using namespace std;

__device__ void cu_strcpy(char* dest, char* src, int dest_index, int src_index) {
	dest[dest_index] = src[src_index];
}

__global__ void produce(char *str, char *res, int *indices) {
	int x = threadIdx.x;

	int index = indices[x];
	for (int i = 0; i < 4 - x; i++) {
		res[index + i] = str[i];
		//cu_strcpy(res, str, index + i, i);
	}
}


int main() {
	int len;

	printf("Enter size of string\n");
	scanf("%d", &len);
	getchar();

	char* str = (char*)malloc(len * sizeof(char));
	char *res = (char*)malloc(len * len * sizeof(char));
	int* indices = (int*)malloc(len * sizeof(int));

	printf("Enter string\n");
	gets_s(str, len);

	memset(indices, 0, len * sizeof(int));
	for (int i = 1; i < len -1; i++) {
		indices[i] = indices[i - 1] + 5 - i;
	}

	char* device_str, * device_res;
	int* device_indices;
	const int int_size = len * sizeof(int);
	const int char_size = len * sizeof(char);

	hipMalloc(&device_str, char_size);
	hipMalloc(&device_res, char_size * len);
	hipMalloc(&device_indices, int_size);

	hipMemcpy(device_str, str, char_size, hipMemcpyHostToDevice);
	hipMemcpy(device_indices, indices, int_size, hipMemcpyHostToDevice );

	produce << <1, len - 1>> > (str, res, indices);

	hipMemcpy(res, device_res, char_size * len, hipMemcpyDeviceToHost);
	
	printf("%s", res);

	return 0;
}