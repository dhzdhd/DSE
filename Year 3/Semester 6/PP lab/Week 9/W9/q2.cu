#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cudart_platform.h"
#include ""
#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <malloc.h>

__global__ void produce(char *str, char *res, int *indices) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;

	int index = indices[x];
	for (int i = 0; i < 4 - x; i++) {
		res[index + i] = str[i];
	}
}

int main() {
	int len;

	printf("Enter size of string\n");
	scanf("%d", &len);
	getchar();

	char* str = (char*)malloc(len * sizeof(char));
	char *res = (char*)malloc(len * len * sizeof(char));
	int* indices = (int*)malloc(len * sizeof(int));

	printf("Enter string\n");
	gets_s(str, len);

	memset(indices, 0, len * sizeof(int));
	for (int i = 1; i < len - 1; i++) {
		indices[i] = indices[i - 1] + len - i;
	}

	char* device_str, * device_res;
	int* device_indices;

	hipMalloc((void **) & device_str, len * sizeof(char));
	hipMalloc((void**)&device_res, len * len * sizeof(char));
	hipMalloc((void**)&device_indices, len * sizeof(int));

	hipMemcpy(device_str, str, len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(device_indices, indices, len * sizeof(int), hipMemcpyHostToDevice);

	produce << <1, len - 1 >> > (device_str, device_res, device_indices);

	hipMemcpy(res, device_res, len * len * sizeof(char), hipMemcpyDeviceToHost);
	
	printf("%s", res);

	hipFree(device_str);
	hipFree(device_res);
	hipFree(device_indices);

	free(str);
	free(res);
	free(indices);

	return 0;
}
